#include "hip/hip_runtime.h"
#include <vector>

#include "gpu_sim.cuh"

void print_gpu_tree_buffer(int *gpu_tree_buffer, unsigned int tree_buffer_size_nodes) {
    unsigned int buffer_size = tree_buffer_size_nodes * NODE_SIZE_BYTES;
    int *host_ptr = (int *)malloc(buffer_size);
    hipMemcpy(host_ptr, gpu_tree_buffer, buffer_size, hipMemcpyDeviceToHost);
    for (int i = 0; i < tree_buffer_size_nodes; i++) {
        for (int j = 0; j < NODE_SIZE_INT; j++) {
            std::cout << host_ptr[i * NODE_SIZE_INT + j] << " ";
        }
        std::cout << buffer_size << std::endl;
    }
    delete host_ptr;
}

void h_clear_tree(int *gpu_tree_buffer, int *used_tree_buffer_size, unsigned int tree_buffer_size_nodes, bool async) {
    dim3 block_dim(32, 1, 1);
    dim3 grid_dim((tree_buffer_size_nodes / block_dim.x) + 1, 1, 1);

    clear_tree<<<grid_dim, block_dim>>>(gpu_tree_buffer, nullptr, tree_buffer_size_nodes);

    if (!async) {
        hipDeviceSynchronize();
    }
}
__global__ void clear_tree(int *tree_buffer, int *used_tree_buffer_size, unsigned int tree_buffer_size_nodes) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < tree_buffer_size_nodes) {
        tree_buffer[tid * NODE_SIZE_INT + TREE_CHILD_OFFSET] = NO_CHILD_NO_PARTICLE;
        tree_buffer[tid * NODE_SIZE_INT + TREE_LOCK_OFFSET] = UNLOCKED;
    }
}

void h_read_tree(int *gpu_tree_buffer, int *gpu_particles_buffer, unsigned int tree_buffer_size_nodes, bool async) {
    dim3 block_dim(32, 1, 1);
    dim3 grid_dim((tree_buffer_size_nodes / block_dim.x) + 1, 1, 1);

    read_tree<<<grid_dim, block_dim>>>(gpu_tree_buffer, gpu_particles_buffer, tree_buffer_size_nodes);

    if (!async) {
        hipDeviceSynchronize();
    }
}
__global__ void read_tree(int *gpu_tree_buffer, int *gpu_particles_buffer, unsigned int tree_buffer_size_nodes) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int tree_buffer_pos = tid * NODE_SIZE_INT;

    if (tid < tree_buffer_size_nodes) {
        // For each leaf node, write the updated particle position back to particle list
        if (gpu_tree_buffer[tree_buffer_pos + TREE_CHILD_OFFSET] == PARTICLE_NO_CHILD && gpu_tree_buffer[tree_buffer_pos + TREE_TYPE_OFFSET] == CANCER_CELL) {
            int particle_id = gpu_tree_buffer[tree_buffer_pos + TREE_ID_OFFSET];
            gpu_particles_buffer[particle_id * PARTICLE_SIZE_INT + PARTICLE_X_OFFSET] = gpu_tree_buffer[tree_buffer_pos + TREE_X_OFFSET];
            gpu_particles_buffer[particle_id * PARTICLE_SIZE_INT + PARTICLE_Y_OFFSET] = gpu_tree_buffer[tree_buffer_pos + TREE_Y_OFFSET];
            gpu_particles_buffer[particle_id * PARTICLE_SIZE_INT + PARTICLE_Z_OFFSET] = gpu_tree_buffer[tree_buffer_pos + TREE_Z_OFFSET];
        }
    }
}

py::array_t<int> walk_particles_gpu(py::array_t<int> initial_particles, py::array_t<int> boundary_particles, int number_of_timesteps, float bound_range, int max_tries, bool random_walk, bool return_gpu_tree_buffer, int tree_buffer_size_nodes) {
    // Create gpu tree buffer
    int *gpu_tree_buffer = nullptr;
    size_t gpu_tree_buffer_size = tree_buffer_size_nodes * NODE_SIZE_BYTES;
    hipMalloc(&gpu_tree_buffer, gpu_tree_buffer_size);

    // Send particle data to the gpu
    size_t particle_count = initial_particles.shape(0);
    int *initial_particles_ptr = static_cast<int *>(initial_particles.request().ptr);
    int *gpu_particles_buffer;
    size_t gpu_particles_buffer_size = particle_count * 3 * sizeof(int);
    hipMalloc(&gpu_particles_buffer, gpu_particles_buffer_size);
    hipMemcpy(gpu_particles_buffer, initial_particles_ptr, gpu_particles_buffer_size, hipMemcpyHostToDevice);

    // Create numpy list to hold result
    std::vector<size_t> shape = {number_of_timesteps * particle_count * 3};
    py::array_t<int> result_array(shape);
    int *result_array_ptr = static_cast<int *>(result_array.request().ptr);

    // Run Kernels for each timestep
    for (int timestep = 0; timestep < number_of_timesteps; timestep++) {
        h_clear_tree(gpu_tree_buffer, nullptr, tree_buffer_size_nodes, true);

        if (random_walk) {
            // Build tree
        }

        h_read_tree(gpu_tree_buffer, gpu_particles_buffer, tree_buffer_size_nodes, true);

        hipDeviceSynchronize();

        // Move data from gpu to host
        int *offset_result_array_ptr = result_array_ptr + (timestep * particle_count * 3);
        hipMemcpy(offset_result_array_ptr, gpu_particles_buffer, gpu_particles_buffer_size, hipMemcpyDeviceToHost);
    }

    // Change windowing for python numpy array
    result_array.resize({(size_t)number_of_timesteps, (size_t)particle_count, (size_t)3});

    // Only used for testing
    if (return_gpu_tree_buffer) {
        std::vector<size_t> shape = {(size_t)tree_buffer_size_nodes * NODE_SIZE_INT};
        py::array_t<int> gpu_tree_result(shape);
        int *gpu_tree_result_ptr = static_cast<int *>(gpu_tree_result.request().ptr);
        hipMemcpy(gpu_tree_result_ptr, gpu_tree_buffer, gpu_tree_buffer_size, hipMemcpyDeviceToHost);
        return gpu_tree_result;
    }

    // Cleanup
    hipFree(gpu_tree_buffer);
    hipFree(gpu_particles_buffer);

    return result_array;
}