#include "hip/hip_runtime.h"
#include <vector>

#include "gpu_sim.cuh"

void print_gpu_tree_buffer(int *gpu_tree_buffer, unsigned int tree_buffer_size_nodes) {
    unsigned int buffer_size = tree_buffer_size_nodes * NODE_SIZE_BYTES;
    int *host_ptr = (int *)malloc(buffer_size);
    hipMemcpy(host_ptr, gpu_tree_buffer, buffer_size, hipMemcpyDeviceToHost);
    for (int i = 0; i < tree_buffer_size_nodes; i++) {
        for (int j = 0; j < NODE_SIZE_INT; j++) {
            std::cout << host_ptr[i * NODE_SIZE_INT + j] << " ";
        }
        std::cout << buffer_size << std::endl;
    }
    delete host_ptr;
}

void h_clear_tree(int *gpu_tree_buffer, int *used_tree_buffer_size, unsigned int tree_buffer_size_nodes, bool async) {
    dim3 block_dim(32, 1, 1);
    dim3 grid_dim((tree_buffer_size_nodes / block_dim.x) + 1, 1, 1);

    clear_tree<<<grid_dim, block_dim>>>(gpu_tree_buffer, used_tree_buffer_size, tree_buffer_size_nodes);

    if (!async) {
        hipDeviceSynchronize();
    }
}
__global__ void clear_tree(int *tree_buffer, int *used_tree_buffer_size, unsigned int tree_buffer_size_nodes) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid == 0) {
        used_tree_buffer_size[0] = NODE_SIZE_INT;
    }

    if (tid < tree_buffer_size_nodes) {
        tree_buffer[tid * NODE_SIZE_INT + TREE_CHILD_OFFSET] = NO_PARTICLE_NO_CHILD;
        tree_buffer[tid * NODE_SIZE_INT + TREE_LOCK_OFFSET] = UNLOCKED;
    }
}

void h_build_tree(int *gpu_tree_buffer, int *used_tree_buffer_size, int *gpu_particles_buffer, unsigned int tree_buffer_size_nodes, int number_of_particles, int particle_type, float bound_range, int max_tries, bool random_walk, bool async) {
    dim3 block_dim(32, 1, 1);
    dim3 grid_dim((number_of_particles / block_dim.x) + 1, 1, 1);

    build_tree<<<grid_dim, block_dim>>>(gpu_tree_buffer, used_tree_buffer_size, gpu_particles_buffer, tree_buffer_size_nodes, number_of_particles, particle_type, bound_range, max_tries, random_walk);

    if (!async) {
        hipDeviceSynchronize();
    }
}
__global__ void build_tree(int *gpu_tree_buffer, int *used_tree_buffer_size, int *gpu_particles_buffer, unsigned int tree_buffer_size_nodes, int number_of_particles, int particle_type, float bound_range, int max_tries, bool random_walk) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < number_of_particles) {
        // State of particle
        bool inserted_particle = false;
        int particle_buffer_pos = tid * PARTICLE_SIZE_INT;
        int original_particle_position[PARTICLE_SIZE_INT];
        int walked_particle_position[PARTICLE_SIZE_INT];
        original_particle_position[PARTICLE_X_OFFSET] = gpu_particles_buffer[particle_buffer_pos + PARTICLE_X_OFFSET];
        original_particle_position[PARTICLE_Y_OFFSET] = gpu_particles_buffer[particle_buffer_pos + PARTICLE_Y_OFFSET];
        original_particle_position[PARTICLE_Z_OFFSET] = gpu_particles_buffer[particle_buffer_pos + PARTICLE_Z_OFFSET];
        int tries_left = max_tries;

        // TODO: Add random
        while (!inserted_particle && tries_left > 0) {
            // Walk the particle (randomize position) (for now write original)
            walked_particle_position[PARTICLE_X_OFFSET] = original_particle_position[PARTICLE_X_OFFSET];
            walked_particle_position[PARTICLE_Y_OFFSET] = original_particle_position[PARTICLE_Y_OFFSET];
            walked_particle_position[PARTICLE_Z_OFFSET] = original_particle_position[PARTICLE_Z_OFFSET];

            // Insert particle into tree
            {
                // Insertion State
                int curr_tree_pos = 0;
                float curr_bound_range = bound_range;
                float bound_start[3];
                bound_start[0] = bound_start[1] = bound_start[2] = -0.5f * bound_range;
                bool completed_insert_attempt = false;

                while (!completed_insert_attempt) {
                    // Travel deeper if non-leaf
                    if (gpu_tree_buffer[curr_tree_pos + TREE_CHILD_OFFSET] >= 0) {
                        completed_insert_attempt = true;
                    }
                    // If leaf get lock and insert here
                    if (UNLOCKED == atomicCAS(&gpu_tree_buffer[curr_tree_pos + TREE_LOCK_OFFSET], UNLOCKED, tid)) {
                        int curr_node_child = gpu_tree_buffer[curr_tree_pos + TREE_CHILD_OFFSET];
                        if (NO_PARTICLE_NO_CHILD == curr_node_child) {
                            gpu_tree_buffer[curr_tree_pos + TREE_ID_OFFSET] = tid;
                            gpu_tree_buffer[curr_tree_pos + TREE_X_OFFSET] = walked_particle_position[PARTICLE_X_OFFSET];
                            gpu_tree_buffer[curr_tree_pos + TREE_Y_OFFSET] = walked_particle_position[PARTICLE_Y_OFFSET];
                            gpu_tree_buffer[curr_tree_pos + TREE_Z_OFFSET] = walked_particle_position[PARTICLE_Z_OFFSET];
                            gpu_tree_buffer[curr_tree_pos + TREE_TYPE_OFFSET] = particle_type;
                            gpu_tree_buffer[curr_tree_pos + TREE_CHILD_OFFSET] = PARTICLE_NO_CHILD;
                            inserted_particle = true;
                        }

                        // Need to move existing particle and new particle into subtree
                        else {
                        }

                        __threadfence();
                        gpu_tree_buffer[curr_tree_pos + TREE_LOCK_OFFSET] = UNLOCKED;
                        completed_insert_attempt = true;
                    }
                }
            }

            // Reset particle for the next iteration
            tries_left--;
        }
    }
}

void h_read_tree(int *gpu_tree_buffer, int *gpu_particles_buffer, unsigned int tree_buffer_size_nodes, bool async) {
    dim3 block_dim(32, 1, 1);
    dim3 grid_dim((tree_buffer_size_nodes / block_dim.x) + 1, 1, 1);

    read_tree<<<grid_dim, block_dim>>>(gpu_tree_buffer, gpu_particles_buffer, tree_buffer_size_nodes);

    if (!async) {
        hipDeviceSynchronize();
    }
}
__global__ void read_tree(int *gpu_tree_buffer, int *gpu_particles_buffer, unsigned int tree_buffer_size_nodes) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int tree_buffer_pos = tid * NODE_SIZE_INT;

    if (tid < tree_buffer_size_nodes) {
        // For each leaf node, write the updated particle position back to particle list
        if (gpu_tree_buffer[tree_buffer_pos + TREE_CHILD_OFFSET] == PARTICLE_NO_CHILD && gpu_tree_buffer[tree_buffer_pos + TREE_TYPE_OFFSET] == CANCER_CELL) {
            int particle_id = gpu_tree_buffer[tree_buffer_pos + TREE_ID_OFFSET];
            gpu_particles_buffer[particle_id * PARTICLE_SIZE_INT + PARTICLE_X_OFFSET] = gpu_tree_buffer[tree_buffer_pos + TREE_X_OFFSET];
            gpu_particles_buffer[particle_id * PARTICLE_SIZE_INT + PARTICLE_Y_OFFSET] = gpu_tree_buffer[tree_buffer_pos + TREE_Y_OFFSET];
            gpu_particles_buffer[particle_id * PARTICLE_SIZE_INT + PARTICLE_Z_OFFSET] = gpu_tree_buffer[tree_buffer_pos + TREE_Z_OFFSET];
        }
    }
}

py::array_t<int> walk_particles_gpu(py::array_t<int> initial_particles, py::array_t<int> boundary_particles, int number_of_timesteps, float bound_range, int max_tries, bool random_walk, bool return_gpu_tree_buffer, int tree_buffer_size_nodes) {
    // Create gpu tree buffer
    int *gpu_tree_buffer = nullptr;
    size_t gpu_tree_buffer_size = tree_buffer_size_nodes * NODE_SIZE_BYTES;
    hipMalloc(&gpu_tree_buffer, gpu_tree_buffer_size);

    // Create single size array to track used space in tree buffer
    int *used_tree_buffer_size = nullptr;
    hipMalloc(&used_tree_buffer_size, sizeof(int));

    // Send particle data to the gpu
    size_t particle_count = initial_particles.shape(0);
    int *initial_particles_ptr = static_cast<int *>(initial_particles.request().ptr);
    int *gpu_particles_buffer;
    size_t gpu_particles_buffer_size = particle_count * 3 * sizeof(int);
    hipMalloc(&gpu_particles_buffer, gpu_particles_buffer_size);
    hipMemcpy(gpu_particles_buffer, initial_particles_ptr, gpu_particles_buffer_size, hipMemcpyHostToDevice);

    // Create numpy list to hold result
    std::vector<size_t> shape = {number_of_timesteps * particle_count * 3};
    py::array_t<int> result_array(shape);
    int *result_array_ptr = static_cast<int *>(result_array.request().ptr);

    // Run Kernels for each timestep
    for (int timestep = 0; timestep < number_of_timesteps; timestep++) {
        h_clear_tree(gpu_tree_buffer, used_tree_buffer_size, tree_buffer_size_nodes, true);
        h_build_tree(gpu_tree_buffer, used_tree_buffer_size, gpu_particles_buffer, tree_buffer_size_nodes, particle_count, CANCER_CELL, bound_range, max_tries, random_walk, true);
        h_read_tree(gpu_tree_buffer, gpu_particles_buffer, tree_buffer_size_nodes, true);

        hipDeviceSynchronize();

        // Move data from gpu to host
        int *offset_result_array_ptr = result_array_ptr + (timestep * particle_count * 3);
        hipMemcpy(offset_result_array_ptr, gpu_particles_buffer, gpu_particles_buffer_size, hipMemcpyDeviceToHost);
    }

    // Change windowing for python numpy array
    result_array.resize({(size_t)number_of_timesteps, (size_t)particle_count, (size_t)3});

    // Only used for testing
    if (return_gpu_tree_buffer) {
        std::vector<size_t> shape = {(size_t)tree_buffer_size_nodes * NODE_SIZE_INT};
        py::array_t<int> gpu_tree_result(shape);
        int *gpu_tree_result_ptr = static_cast<int *>(gpu_tree_result.request().ptr);
        hipMemcpy(gpu_tree_result_ptr, gpu_tree_buffer, gpu_tree_buffer_size, hipMemcpyDeviceToHost);
        return gpu_tree_result;
    }

    // Cleanup
    hipFree(gpu_tree_buffer);
    hipFree(gpu_particles_buffer);

    return result_array;
}