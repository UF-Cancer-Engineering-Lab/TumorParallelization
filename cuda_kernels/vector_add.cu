
#include <hip/hip_runtime.h>
extern "C" __device__ void print_hi()
{   
    printf("HI!");
}

extern "C" __global__ void vector_add(float *a, float *b, float *c)
{   
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    c[tid] = a[tid] + b[tid];
    print_hi();
}